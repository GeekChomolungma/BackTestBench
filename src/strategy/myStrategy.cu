// includes, system
#include <stdio.h>
#include <vector>
#include <dtos/kline.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// indicator lib
#include "indicator/ta.cuh"

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper utility functions

void kernel_wrapper(int argc, const char* argv[], std::vector<Kline>& rawData) {
    int devID;
    hipDeviceProp_t deviceProps;

    printf("[%s] - Starting...\n", argv[0]);
    printf("kline size is %zd\n", rawData.size());
    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char**)argv);

    // get device name
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s]\n", deviceProps.name);

    // allocate host memeory
    std::vector<Kline> hostSrc = rawData;
    size_t n = rawData.size();
    size_t nbytes = rawData.size() * sizeof(Kline);
    printf("rawData size is %d, nbytes is %d \n", n, nbytes);

    // allocate device memory
    Kline* deviceRaw = 0;
    float* deviceEma = 0;
    checkCudaErrors(hipMalloc((void**)&deviceRaw, nbytes));
    checkCudaErrors(hipMalloc((void**)&deviceEma, n*sizeof(float)));
    checkCudaErrors(hipMemset(deviceRaw, 255, nbytes));
    checkCudaErrors(hipMemset(deviceRaw, 0, n * sizeof(float)));

    // set kernel launch configuration
    dim3 threads = dim3(32, 1);
    dim3 blocks = dim3(n / threads.x + 1, 1);

    // create cuda event handles
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    StopWatchInterface* timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);

    checkCudaErrors(hipDeviceSynchronize());
    float gpu_time = 0.0f;

    // asynchronously issue work to the GPU (all to stream 0)
    checkCudaErrors(hipProfilerStart());
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);
    hipMemcpyAsync(deviceRaw, rawData.data(), nbytes, hipMemcpyHostToDevice, 0);
    test_kernel <<<blocks, threads, 0, 0 >>> (deviceRaw, deviceEma,1, n, 10, 0.2);
    hipMemcpyAsync(rawData.data(), deviceRaw, nbytes, hipMemcpyDeviceToHost, 0);
    std::vector<float> hostEma(n);
    hipMemcpyAsync(hostEma.data(), deviceEma, n * sizeof(float), hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);
    checkCudaErrors(hipProfilerStop());

    //// have CPU do some work while waiting for stage 1 to finish
    //unsigned long int counter = 0;

    //while (hipEventQuery(stop) == hipErrorNotReady) {
    //    counter++;
    //}

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f ms\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f ms\n", sdkGetTimerValue(&timer));
    printf("EMA in host length is: %d, the 0th EMA is: %f, the 1th EMA is: %f \n", hostEma.size(), hostEma[0], hostEma[1]);

    // release resources
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    //checkCudaErrors(hipHostFree(hostSrc));
    checkCudaErrors(hipFree(deviceRaw));
    checkCudaErrors(hipFree(deviceEma));

    return;
   /* exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);*/
}