// includes, system
#include <stdio.h>
#include <vector>
#include <utility> // for std::pair
#include <dtos/kline.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// indicator lib
#include "indicator/ta.cuh"

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper utility functions

void kernel_wrapper(int argc, const char* argv[], std::vector<Kline>& rawData, std::vector<std::pair<int, int>>& dataIndexes) {
    int devID;
    hipDeviceProp_t deviceProps;

    printf("[%s] - Starting...\n", argv[0]);
    printf("kline size is %zd\n", rawData.size());
    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char**)argv);

    // get device name
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s]\n", deviceProps.name);

    // allocate host memeory
    std::vector<Kline> hostSrc = rawData;
    size_t n = rawData.size();
    size_t nbytes = rawData.size() * sizeof(Kline);
    printf("rawData size is %zd, nbytes is %zd \n", n, nbytes);

    // calculate stock number and 
    // get start and end index of each stock
    size_t stockNumber = dataIndexes.size();
    int* startIndexes = new int[stockNumber];
    int* endIndexes = new int[stockNumber];
    for (auto i = 0; i < stockNumber; i++) {
        startIndexes[i] = dataIndexes[i].first;
        endIndexes[i] = dataIndexes[i].second;
    }

    // allocate device memory
    Kline* deviceRaw = 0;
    float* deviceEma = 0;
    int* deviceStartInd = 0;
    int* deviceEndInd = 0;
    checkCudaErrors(hipMalloc((void**)&deviceRaw, nbytes));
    checkCudaErrors(hipMalloc((void**)&deviceEma, n*sizeof(float)));
    checkCudaErrors(hipMemset(deviceRaw, 255, nbytes));
    checkCudaErrors(hipMemset(deviceRaw, 0, n * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&deviceStartInd, stockNumber*sizeof(int)));
    checkCudaErrors(hipMemset(deviceStartInd, 0, stockNumber * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&deviceEndInd, stockNumber * sizeof(int)));
    checkCudaErrors(hipMemset(deviceEndInd, 0, stockNumber * sizeof(int)));

    // set kernel launch configuration
    dim3 threads = dim3(32, 1);
    dim3 blocks = dim3(n / threads.x + 1, 1);

    // create cuda event handles
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    StopWatchInterface* timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);

    checkCudaErrors(hipDeviceSynchronize());
    float gpu_time = 0.0f;

    // new a stream for this task
    hipStream_t stream;
    hipStreamCreate(&stream);

    // asynchronously issue work to the GPU (all to stream 0)
    checkCudaErrors(hipProfilerStart());
    sdkStartTimer(&timer);
    hipEventRecord(start, stream);
    hipMemcpyAsync(deviceRaw, rawData.data(), nbytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(deviceStartInd, startIndexes, stockNumber * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(deviceEndInd, endIndexes, stockNumber * sizeof(int), hipMemcpyHostToDevice, stream);
    test_kernel <<<blocks, threads, 0, stream >>> (deviceRaw, deviceEma, stockNumber, deviceStartInd, deviceEndInd, 10, 0.2);
    hipMemcpyAsync(rawData.data(), deviceRaw, nbytes, hipMemcpyDeviceToHost, stream);
    std::vector<float> hostEma(n);
    hipMemcpyAsync(hostEma.data(), deviceEma, n * sizeof(float), hipMemcpyDeviceToHost, stream);
    hipEventRecord(stop, stream);
    sdkStopTimer(&timer);
    checkCudaErrors(hipProfilerStop());

    //// have CPU do some work while waiting for stage 1 to finish
    //unsigned long int counter = 0;

    //while (hipEventQuery(stop) == hipErrorNotReady) {
    //    counter++;
    //}

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f ms\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f ms\n", sdkGetTimerValue(&timer));
    printf("EMA in host length is: %zd, the 0th EMA is: %f, the 1th EMA is: %f \n", hostEma.size(), hostEma[0], hostEma[1]);

    // release resources
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    //checkCudaErrors(hipHostFree(hostSrc));
    checkCudaErrors(hipFree(deviceRaw));
    checkCudaErrors(hipFree(deviceEma));
    checkCudaErrors(hipFree(deviceStartInd));
    checkCudaErrors(hipFree(deviceEndInd));
    hipStreamDestroy(stream);
    delete[] startIndexes;
    delete[] endIndexes;

    return;
   /* exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);*/
}